#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>
#include <chrono>

// Efektywny dost�p do danych
__global__ void computeSumKernelEfficient(const float* TAB, float* OUT, int N, int R, int OUT_size) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < OUT_size && j < OUT_size) {
        float sum = 0.0f;
        for (int di = -R; di <= R; ++di) {
            for (int dj = -R; dj <= R; ++dj) {
                int ni = i + R + di;
                int nj = j + R + dj;
                if (ni >= 0 && ni < N && nj >= 0 && nj < N) {
                    sum += TAB[ni * N + nj];
                }
            }
        }
        OUT[i * OUT_size + j] = sum;
    }
}

__global__ void computeSumKernelInefficient(const float* TAB, float* OUT, int N, int R, int OUT_size) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < OUT_size && j < OUT_size) {
        float sum = 0.0f;
        for (int di = -R; di <= R; ++di) {
            for (int dj = -R; dj <= R; ++dj) {
                // Efektywny dost�p: przeskakiwanie przez elementy
                int ni = i + R + di;
                int nj = j + R + dj;
                if (ni >= 0 && ni < N && nj >= 0 && nj < N) {
                    sum += TAB[ni * N + nj];
                }
            }
        }
        OUT[i * OUT_size + j] = sum;
    }
}


hipError_t computeSumGPU(const float* TAB, float* OUT, int N, int R, int BS, bool efficient, int k) {
    int OUT_size = N - 2 * R;
    size_t size_TAB = N * N * sizeof(float);
    size_t size_OUT = OUT_size * OUT_size * sizeof(float);

    float* d_TAB = nullptr;
    float* d_OUT = nullptr;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&d_TAB, size_TAB);
    if (cudaStatus != hipSuccess) return cudaStatus;

    cudaStatus = hipMalloc((void**)&d_OUT, size_OUT);
    if (cudaStatus != hipSuccess) {
        hipFree(d_TAB);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(d_TAB, TAB, size_TAB, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        hipFree(d_TAB);
        hipFree(d_OUT);
        return cudaStatus;
    }

    dim3 threadsPerBlock(BS, BS);
    dim3 blocksPerGrid((OUT_size + BS - 1) / BS, (OUT_size + BS - 1) / BS);

    if (efficient) {
        computeSumKernelEfficient << <blocksPerGrid, threadsPerBlock >> > (d_TAB, d_OUT, N, R, OUT_size);
    }
    else {
        computeSumKernelInefficient << <blocksPerGrid, threadsPerBlock >> > (d_TAB, d_OUT, N, R, OUT_size);
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        hipFree(d_TAB);
        hipFree(d_OUT);
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        hipFree(d_TAB);
        hipFree(d_OUT);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(OUT, d_OUT, size_OUT, hipMemcpyDeviceToHost);

    hipFree(d_TAB);
    hipFree(d_OUT);

    return cudaStatus;
}

